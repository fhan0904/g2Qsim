#include "hip/hip_runtime.h"
/* kernel routine starts with keyword __global__ */

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

// CUDA includes
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// Root includes
#include <TFile.h>
#include <TH1.h>
#include <TH2.h>

// calo parameters
#define NXSEG 9
#define NYSEG 6
#define NSEG 54

// Qmethod parameters
#define TIMEDECIMATION 16

// simulator parameters
#define NEMAX 5000 // max electrons per fill per calo
#define NTMAX 6    // max threshold histograms

// output root file
TFile *f;

// output histograms
TH1D *hHits1D, *hEnergy1D; // diagnostics histograms of truth hit time, energy distributions
TH1D *hFlush1D[NSEG*NTMAX], *hFlush1Dlost[NSEG*NTMAX]; // per xtal, 1D q-method time distributions (above / below threshold setting) 
TH2D *hFlush2D[NSEG], *hFlush2DCoarse[NSEG]; // per xtal, 2D q-method time distribution
TH2D *hFlush2DSum, *hFlush2DCoarseSum; // per calo, 2D q-method time distribution

float toddiff(struct timeval*, struct timeval*);

// function for calculation time intervals                                                               
float toddiff(struct timeval *tod2, struct timeval *tod1) {
  float fdt, fmudt;
  long long t1, t2, mut1, mut2;
  long long dt, mudt;

  t1 = tod1->tv_sec;
  mut1 = tod1->tv_usec;
  t2 = tod2->tv_sec;
  mut2 = tod2->tv_usec;
  dt = ( t2 - t1);
  mudt = ( mut2 - mut1);
  fdt = (float)dt;
  fmudt = (float)mudt;

  return fdt + 1.0e-6*fmudt;
}

/* 
   GPU kernel function to initialize the random states.
Each thread gets same seed, a different sequence number, 
and no offset 
*/
__global__ void init_rand( hiprandState *state, unsigned long long offset, unsigned long long seed) {

   // thread index
   int idx = blockIdx.x*256 + threadIdx.x;

   hiprand_init( seed, idx, 0, &state[idx]);
}

/*
GPU kernel utility function to initialize fill/flush data arrays
*/
__global__ void zero_int_array( int32_t *array, int length) {

   // thread index
  int idx = blockIdx.x*256 + threadIdx.x;

  if (idx < length) *(array + idx) = 0;
}

/*
GPU kernel utility function to initialize fill/flush data arrays
*/
__global__ void zero_float_array( float *array, int length) {

   // thread index
  int idx = blockIdx.x*256 + threadIdx.x;
  if (idx < length) {
    *(array + idx) = 0.0;
  }
}

/*
GPU kernel user function to build uniform time distribution
*/
__global__ void make_rand( hiprandState *state, float *randArray) {

   // thread index
   int idx = blockIdx.x*256 + threadIdx.x;

   hiprandState localState = state[idx];
   randArray[idx] = hiprand_uniform( &localState);
   state[idx] = localState;
}

/*
GPU kernel user function to build decay curve distribution
*/
__global__ void make_randexp( hiprandState *state, float *randArray, float tau) {

   // thread index
   int idx = blockIdx.x*256 + threadIdx.x;

   hiprandState localState = state[idx];
   randArray[idx] = -tau * log( 1.0 -hiprand_uniform(&localState) );
   state[idx] = localState;
}

/*
GPU kernel user function to build each fills time distribution
*/
__global__ void make_randfill( hiprandState *state, int32_t *hitArray, int32_t *fillArray, float *hitSumArray, float *fillSumArray,  float *energySumArray, int ne, int fill_buffer_max_length, int nfills, bool fillnoise) {

  // single thread make complete fill with ne electrons

   const float tau = 6.4e4;                              // muon time-dilated lifetime (ns)
   const float omega_a = 1.438e-3;                       // muon anomalous precession frequency (rad/ns)
   const float magicgamma = 29.3;                        // gamma factor for magic momentum 3.094 GeV/c
   const int GeVToADC = 2048./6.2;                       // energy-ADC counts conversion (ADC counts / energy GeV)
   const int nsPerTick = TIMEDECIMATION*1000/800;        // Q-method histogram bin size (ns), accounts for 800MHz sampling rate
   const float Elab_max = 3.1;                           // GeV, maximum positron lab energy
   const float Pi = 3.1415926;                           // Pi
   const float cyclotronperiod = 149.0/nsPerTick;        // cyclotron period in histogram bin units
   const float anomalousperiod = 4370./nsPerTick;        // anomalous period omega_c-omega_s in histogram bin units
   const int nxseg = NXSEG, nyseg = NYSEG, nsegs = NSEG; // calorimeter segmentation

   // parameters for empirical calculation of positron drift time calculation from energy via polynomial
   float p0 =    -0.255134;
   float p1 =      65.3034;
   float p2 =     -705.492;
   float p3 =      5267.21;
   float p4 =     -23986.5;
   float p5 =      68348.1;
   float p6 =      -121761;
   float p7 =       131393;
   float p8 =       -78343;
   float p9 =      19774.1;

   // variables for muon decay / kinematics 
   float t, y, A, n;   // mu-decay parameters
   float r, r_test;    // mu decay rate 

   // thread index
   int idx = blockIdx.x*256 + threadIdx.x;

   // one thread per fill
   if (idx < nfills) {

     // state index for random number generator
     hiprandState localState = state[idx];
     
     // make noise for each fill if fillnoise true (time consuming)
     if (fillnoise) {

       float pedestal = 0., sigma = 4.; // paramters for noise distribution

       int32_t noise; 
       for (int i = 0; i < nsegs*fill_buffer_max_length; i++){

	 noise = pedestal + sigma * hiprand_normal(&localState); // generate Gaussian noise using normal distribution
         atomicAdd( &(fillSumArray[ i ]), (float)noise );       // add fill-by-fill noise to flush buffer
       }
     }
     
     int nhit = 0; // good hit counter
     float theta = 0; // decay angle
     
     // parameters for positron x,y, time
     float xrand, yrand, xmax; // x,y coordinate random numbers and endpoint of hit distribution across calo x-coordinate
     
     // parameters for calculating the positron drift time
     float ylab, phase, drifttime; 

     // paraters for positron time, ADC counts, and x/y coordinates
     float tick, ADC, xcoord, ycoord; 
     
     // arrays for storing the hit information before time-ordering (ADCnorm is used for pile-up correction)
     float tickstore[NEMAX], ADCstore[NEMAX], xcoordstore[NEMAX], ycoordstore[NEMAX];
     int  iold[NEMAX];

     // find hit times, energies, x-coordinate, y-coordinate for ne generated electrons from muon decay
     while (nhit < ne){ // should randomize the hits per fill

       // Get muon decay time     
       t = -tau * log( 1.0 - hiprand_uniform(&localState) );     // random from exp(-t/tau) using uniform random number 0->1
       tick = t/nsPerTick;                                      // convert from ns to Q-method histogram bins
       if ( ( (int)tick ) >= fill_buffer_max_length ) continue; // time out-of-bounds 
       
       // Get positron lab energy. Obtained by generating the position energy, angle distribution in muon rest frame
       y = hiprand_uniform(&localState);
       A = (2.0*y - 1)/(3.0 - 2.0*y);
       n = y*y*(3.0 - 2.0*y);
       r_test = n*(1.0-A*cos(omega_a*t))*0.5;
       r = hiprand_uniform(&localState);  
       if ( r >= r_test ) continue;

       theta = Pi*hiprand_uniform(&localState);  // check me
       float Elab = 0.5 *Elab_max * y * ( 1.0 + cos(theta));  // boost to lab frame

       // Account for acceptance of calorimeter using empirical, energy-dependent calo acceptance
       // for now a very simple empirical acceptance, zero below ElabMin, unit above ElabMin
       float ElabMin = 0.5;
       ///if (Elab < ElabMin) continue;

       // Variable ADC is total ADC samples of positron signal at 800 MMz sampling rate with 6.2 GeV max range over 2048 ADC counts
       ADC = GeVToADC*Elab; 

       // Divide by maximum fraction of positron signal in single 800 MHz bin (is ~0.4 from erfunc plot of 5ns FWHM pulse 
       // in peak sample at 800 MHz sampling rate
       ADC = ADC/0.4; 

       // Add empirical energy-dependent drift time, see https://muon.npl.washington.edu/elog/g2/Simulation/229 
       // using empirical distribution for relation between energy and time
       ylab = Elab/Elab_max;
       phase = p0 + p1*ylab + p2*ylab*ylab + p3*ylab*ylab*ylab + p4*ylab*ylab*ylab*ylab 
	 + p5*ylab*ylab*ylab*ylab*ylab + p6*ylab*ylab*ylab*ylab*ylab*ylab + p7*ylab*ylab*ylab*ylab*ylab*ylab*ylab 
	 + p8*ylab*ylab*ylab*ylab*ylab*ylab*ylab*ylab + p9*ylab*ylab*ylab*ylab*ylab*ylab*ylab*ylab*ylab; // phase in mSR units of omega_a
       drifttime = anomalousperiod * phase / (2.*Pi*1000.); // convert the omega_a phase to drift time in Q-method histogram bin units

       tick = tick + drifttime;
       
       // generate the x, y coordinates of positron hit on calorimeter

       // simple random (x, y) coordinates
       //xcoord = nxseg * hiprand_uniform(&localState);
       //ycoord = nyseg * hiprand_uniform(&localState);
       
       // make rough empirical x-distribution obtained from  https://muon.npl.washington.edu/elog/g2/Simulation/258 (Robin)
       // and rough empirical y-distribution obtained from  https://muon.npl.washington.edu/elog/g2/Simulation/256 (Pete)
       if ( ylab > 0.7 ) {
	 xmax = 185.-533.3*(ylab-0.7);
       } else {
	 xmax = 185.;
       }
       xrand = hiprand_uniform(&localState);
       xcoord = xmax*xrand/25.0; // x-coordinate -> mm -> segments
       yrand = hiprand_uniform(&localState);
       ycoord = 1.0+(nyseg-2.0)*yrand; // y-coordinate -> segments

       // q-method histogram bin from decay time in bin units
       int itick = (int)tick;

       // hit arrays are arrays of xtal-summed calo hits (not individual xtal hits) for diagnostics
       // if using hitSumArray flush buffer
       atomicAdd( &(hitSumArray[ itick ]), 1.0);
       // if using hitArray flush buffer
       //hitArray[ itick + fill_buffer_max_length*idx ]++; 

       // energy arrays of true energy of each hit (not individual xtal energies) for diagnostics
       // if using energySumArray flush buffer
       atomicAdd( &(energySumArray[ (int)ADC ]), 1.0);
       // if using hitArray flush buffer
       //hitArray[ ADC + energybins*idx ]++; 
       
       // put hit information (time, ADC counts, x/y coordinates, hit index) into hit array
       // used in time-ordering the hits that's needed for applying pile-up effects
       tickstore[nhit] = tick;
       ADCstore[nhit] = ADC;
       xcoordstore[nhit] = xcoord;
       ycoordstore[nhit] = ycoord;
       iold[nhit] = nhit;
       nhit++;
     }
     //printf("fill %i, hits %i\n", idx, nhit);
     
     // sort array of positron hits into ascending time-order
     int itemp;
     float temp;
     for (int i = 0; i < nhit; ++i) {
       for (int j = i + 1; j < nhit; ++j) {
	 // if higher index array element j is earlier (t_j < t_i) than lower index  array element i then swap elements
	 if (tickstore[i] > tickstore[j]) {
	   // swap times if hit i is later than hit j
	   temp = tickstore[i]; 
	   tickstore[i] = tickstore[j];
	   tickstore[j] = temp;
	   // swap indexes if hit i is later than hit j for later use in swapping ADC, x, y coordinates
	   itemp = iold[i]; 
	   iold[i] = iold[j];
	   iold[j] = itemp;
	 }
       }
     }

     // simple test of pileup effect - doesn't handle the segmentation and effect of amplitude of prior pulse
     // short-term gain-change time constant 30ns, amplitude 4%/1000 
     // long-term gain-change time constant 10us, amplitude 0.012/(1000*(1-exp(-10./64.)) ~ 10^-4 
     // see SIPM paper https://arxiv.org/pdf/1611.03180.pdf
     //float tauG = 10000.0/nsPerTick, ampG = 0.001;
     /*
     for (int j = 1; j < nhit; j++){
       for (int k = 0; k < j; k++){

     	float dt = tickstore[j] - tickstore[k];
	ADCstore[iold[j]] *= 1.0 - ampG*exp(-dt/tauG); 
       }
     }
     */

     // parameters for distributing the ADC counts over calo xtals.
     // parameters for empirical Gaussian distribution of energy across neighboring segments. Used 
     // https://muon.npl.washington.edu/elog/g2/SLAC+Test+Beam+2016/260 and position where energy in 
     // neighboring xtal is 16% (1 sigma) - giving sigma = 0.19 in units of crystal size 
     //float xsig = 0.01, ysig = 0.01; // test with very small spread
     //float xsig = 0.5, ysig = 0.5; // test with very large spread
     float xsig = 0.19, ysig = 0.19; // xtal size units
     
     // parameters for distributing the ADC counts over time bins of q-method histogram
     // approx sigma width of 2.1ns from https://muon.npl.washington.edu/elog/g2/SLAC+Test+Beam+2016/38
     //const float width = 0.21/nsPerTick; // test - make pulse width x10 smaller
     //const float width = 21.0/nsPerTick; // test  - make pulse width x10 larger
     const float width = 2.1/nsPerTick; // pulse sigma in q-method bin width units

     // parameters for pile-up effects
     // simple time constant, pulse amplitude and normalization paramter of pileup effect of prior pulses
     float tauG = 30.0/nsPerTick;
     float ampG = 0.04;
     float ADCnorm = 812;

     float ADCstoresegment[54][NEMAX]; // array used for xtal-by-xtal pileup effects

     // loop over time-ordered positron hits
     for (int i = 0; i < nhit; i++){
       
       // time array is already time-ordered
       tick = tickstore[i]; 
       // other arrays aren't already time-ordered
       ADC = ADCstore[iold[i]];
       xcoord = xcoordstore[iold[i]];
       ycoord = ycoordstore[iold[i]];

       // itick is bin of q-method time histogram
       // rtick is time within bin of q-method time histogram
       int itick = (int)tick;
       float rtick = tick - itick;

       // loop over the array of xtals and distribute the total ADC counts (ADC) to each xtal (ADC segment)
       // using the hit coordinates xcoord, ycoord and distribution paramters xsig, ysig. 
       float fsegmentsum = 0.0; // diagnostic parameter for distribution of energy over segments
       for (int ix = 0; ix < nxseg; ix++) {
	 for (int iy = 0; iy < nyseg; iy++) {
	   
	   // energy in segment (assume a Gaussian distribution about xc, yc
           float fsegmentx = 0.5*(-erfcf((ix+1.0-xcoord)/(sqrt(2.)*xsig))+erfcf((ix-xcoord)/(sqrt(2.)*xsig)));
	   float fsegmenty = 0.5*(-erfcf((iy+1.0-ycoord)/(sqrt(2.)*ysig))+erfcf((iy-ycoord)/(sqrt(2.)*ysig)));
           float fsegment = fsegmentx*fsegmenty;
	   float ADCsegment = fsegment*ADC;
           fsegmentsum += fsegment;

	   if (ADCsegment < 1.0) continue; // avoid pileup calc if signal in xtal is neglibible
           
           // handle pulse-pileup on segment-to-segment basis

	   /* xtal-by-xtal pileup calculation

	   // store ADC counts of "fired" xtal
	   ADCstoresegment[ix+iy*nxseg][i] = ADCsegment; // store samples of "fired" segment

           // handle pileup correction on segment-by-segment basis by looping over prior hits
           // uses pileup parameters ampG, tauG
	   for (int ipu = 0; ipu < i; ipu++) {
	     float dt = tickstore[i] - tickstore[ipu];
	     ADCsegment *= 1.0 - (ADCstoresegment[ix+iy*nxseg][i]/ADCnorm)*ampG*exp(-dt/tauG);
	   }

	   */

	   // offset needed for storing xtal hits in samples array
	   int xysegmentoffset = (ix+iy*nxseg)*fill_buffer_max_length; 
	   
	   // do time smearing of positron pulse over several contiguous time bins
           // just loop over bins k-1, k, k+1 as negligible effect for other bins   
	   float tfracsum = 0.0; // diagnostic for distribution of energy over segments
	   for (int k=-1; k<=1; k++) {

	     int kk = k + itick;
	     if ( kk < 0 || kk >= fill_buffer_max_length ) continue;

              // energy in bin (assume a Gaussian distribution about tick (time within central bin)
	     float tfrac = 0.5*(-erfcf((kk+1.0-tick)/(sqrt(2.)*width))+erfcf((kk-tick)/(sqrt(2.)*width)));
             float ADCfrac = ADCsegment*tfrac; 
	     tfracsum += tfrac;

	     if ( ADCfrac > 2048 ) ADCfrac = 2048; // apply overflow of ADC counts

             // if using fillArray fill buffer
	     //if ( ADCfrac >= 1 ) *(fillArray + nsegs*fill_buffer_max_length*idx + xysegmentoffset + kk ) += ADCfrac;  // fill buffer
             // if using fillSumArray flush buffer
	     if ( ADCsegment >= 1 ) atomicAdd( &(fillSumArray[ xysegmentoffset + kk ]), ADCfrac );

	   } // end of time smearing

           // for no time smearing all xtal ADC counts in single time bin
	   //atomicAdd( &(fillSumArray[ xysegmentoffset + itick ]), (float)ADCsegment );

           // simple-minded testing of effects of tail of SiPM pulse
           // e.g. the problem with long tails on SiPM pulses from AC coupling

	   /*
	   int maxtaillength = 2000; // max tail length calculation  in q-method hostigrma bins
           float tailamp = -1.0e-3, tailtail = 1000.; // pulse tail parameters

	   for (int k = 0; k <= maxtaillength; k++) {
	     int kk = k + itick;

	     if ( kk < 0 || kk >= fill_buffer_max_length ) continue;

	     // energy in bin (assume a Gaussian distribution about tick
	     float tfrac = tailamp*exp(-float(k)/tailtau); // exponential tail
             float ADCfrac = ADCsegment*tfrac; // need to add some noise
	     tfracsum += tfrac;

	     if ( ADCfrac > 2048 ) ADCfrac = 2048;

             // if using fillArray fill buffer
	     //if ( ADCfrac >= 1 ) *(fillArray + nsegs*fill_buffer_max_length*idx + xysegmentoffset + kk ) += ADCfrac;  // fill buffer
             // if using fillSumArray flush buffer
	     if ( ADCsegment >= 1 ) atomicAdd( &(fillSumArray[ xysegmentoffset + kk ]), ADCfrac );

	   } // end of ADC taile
	   */

	 } // y-distribution loop
       } // x-distribution loop

     } // time-ordered hits hits
     
     // state index for random number generator
     state[idx] =  localState;
     
     /*
     // fill pattern for testing 
     for (int i = 0; i < nsegs; i++) {
     
     for (int j = 0; j < fill_buffer_max_length; j++) {
     
     *(fillArray + nsegs*fill_buffer_max_length*idx + fill_buffer_max_length*i + j) += i;  // fill buffer
     }
     }
     */
     
   } // enf of if idx < nfills
}

/*
GPU kernel function - builds fillSumArray from fillArray if fillArray is used and introduces noise at flush-level
*/
__global__ void make_fillsum( hiprandState *state, int32_t *fillArray, float *fillSumArray, int nfills, int fill_buffer_max_length, bool flushnoise ) {

  // thread index
  int idx = blockIdx.x*256 + threadIdx.x;
  hiprandState localState = state[idx];

  int nxsegs = NXSEG, nysegs = NYSEG, nsegs = NSEG;
  
  // fill_buffer_max_length is Q-method bins per segment per fill
  if (idx < nsegs * fill_buffer_max_length) {

    // add all the fills in flush
    for (int i = 0; i < nfills; i++) {
      *(fillSumArray + idx) += (float) *(fillArray + (nsegs*fill_buffer_max_length)*i + idx );  // fill buffer
    }

    // add noise at flush level
    if (flushnoise) {
      float pedestal = 0., sigma = 4.; // parameters of noise at flush level
      float noise = pedestal + sigma * hiprand_normal(&localState); // random from Gaussian using uniform random number 0->1
      *(fillSumArray + idx) += noise;  // fill buffer
      state[idx] = localState;
    }
 
  }
}

/*
GPU kernel function - builds hitSumArray from hitArray if hitArray is used
*/
__global__ void make_hitsum( int32_t *hitArray, float *hitSumArray, int nfills, int fill_buffer_max_length, bool flushnoise) {

  int idx = blockIdx.x*256 + threadIdx.x;
  
  // fill_buffer_max_length is Q-method bins per segment per fill
  if (idx < fill_buffer_max_length) {
    
    // initialize flush to zero (now use hipMemset)
    //  *(hitSumArray + idx) = 0.0;
    
    // add fills in flush
    for (int i = 0; i < nfills; i++) {
      *(hitSumArray + idx) += (float) *( hitArray + (fill_buffer_max_length)*i + idx );  // fill buffer
    }   
  }
}


/*
GPU kernel function - builds energySumArray from energuArray if energyArray is used
*/
__global__ void make_energysum( int32_t *energyArray, float *energySumArray, int nfills, int fill_buffer_max_length, bool flushnoise) {

  int idx = blockIdx.x*256 + threadIdx.x;
  
  // fill_buffer_max_length is Q-method bins per segment per fill
  if (idx < fill_buffer_max_length) {
    
    // initialize flush to zero (now use hipMemset)
    //  *(energySumArray + idx) = 0.0;
    
    // add fills in flush
    for (int i = 0; i < nfills; i++) {
      *(energySumArray + idx) += (float) *( energyArray + (fill_buffer_max_length)*i + idx );  // fill buffer
    }   
  }
}

/* 
main program

usage
./rand ne nfills nflushes threshold 

where arguments are number of electrons in fill, number of fills in flush, number of flushes in run, and threshold
applied at flush level

*/
int main(int argc, char * argv[]){

  hipError_t err;
 
  // define nthreads, nblocks, arrays for GPU
  int nthreads = 256, nblocks1, nblocks2, nblocks3, nblocks4, nblocks5;

  // parameters for number of electrons in fill, number of fills in flush, and flushes in run
  int ne, nfills, nflushes, nthresholds;
  // parameter for qmethod threshold, Set to -999 for zero-threshold qmethod. this threshold is applied to flushes not fills 
  float threshold;

  // for state of randum generators
  hiprandState *d_state, *d_state2;
  float *h_randArray, *d_randArray;

  // Q-method arrays 
  int32_t *h_fillArray, *d_fillArray;
  float *h_fillSumArray, *d_fillSumArray; 
  // for hits arrays 
  int32_t *h_hitArray, *d_hitArray;
  float *h_hitSumArray, *d_hitSumArray; 
  // for energy array
  int32_t *h_energyArray, *d_energyArray;
  float *h_energySumArray, *d_energySumArray; 

  // define fill length, clock tick for simulation
  //const int nsPerFill = 4096, nsPerTick = 16; 
  //const int nsPerFill = 560000, nsPerTick = 16; 
  const int nsPerFill = 560000, nsPerTick = TIMEDECIMATION*1000/800; 

  int fill_buffer_max_length = nsPerFill / nsPerTick; // fill length in unit of hostogram bins
  int nxsegs = NXSEG, nysegs = NYSEG, nsegs = NSEG; // calo segmentation parameters
  int energybins = 4096; // number of energy histogram bins
 
  // define run, flush, fill structure from command line arguments
  printf("number of argurments of command %i\n", argc); 
  if (argc == 1) {
    ne = 5500;   // on average 1100 good e's per fill, 5500 e's per fill
    nfills = 256;
    nflushes = 1;
    threshold = -999.; // -999. for zero-threashold qmethod
    nthresholds = 1; // number of steps of threshold
  } else {
    ne = atoi(argv[1]);
    nfills = atoi(argv[2]);
    nflushes = atoi(argv[3]);
    threshold = atoi(argv[4]);
    nthresholds = atoi(argv[5]);
  }

  // from TDR 
  // events for 100ppb stat uncertainty, 1.6e11
  // >30us, >1.86 GeV positrons for 24 calos per fill, 1100
  // positrons for 24 calos per fill, 5500 (from TDR Fig 16.8 for energy and exp(-t/tau) for time) 
  // fills for 140 ppb, 1.5e8
  // from elog
  // we need 1.6e11 events to reach 100ppb statistical precision. (TDR page 119)
  // will acquire the 1.6x10e11 events in 1.5e8 fills (TDR table 5.1, pg 122)  
  // i.e 1100 >1.8 GeV events per fill (TDR page 122) (note this is higher than the 700 >1.8 GeV events per fill in (docdb 676)
  // x1.6 (TCut ? no flash) x 2.4 (E cut) =  4200 total events per fill, 180 total events per calo per fill (docdb 676)
 
  // divide by 24 for per calorimeter rate. 
  ne = ne / 24;

  printf("nsegments per calo %d, ne per fill per calo %d, nfills per flush %d, nflushes per run %d, flush-level threshold interval %f, threshold number %d\n", 
	 nsegs, ne, nfills, nflushes, threshold, nthresholds);

  // define grid structure for run, flush, fill structure 
  nblocks1 = nfills / nthreads + 1;
  nblocks2 = ( nsegs * fill_buffer_max_length + nthreads - 1 )/ nthreads;
  nblocks3 = ( fill_buffer_max_length + nthreads - 1 )/ nthreads;
  nblocks4 = ( nsegs * nfills * fill_buffer_max_length + nthreads - 1 )/ nthreads;
  nblocks5 = ( nfills * fill_buffer_max_length + nthreads - 1 )/ nthreads;
  printf("per flush grid: nthreads %i, nblocks %i nthreads*nblocks %i\n", nthreads, nblocks1, nthreads*nblocks1 );
  printf("per bin grid: nthreads %i, nblocks %i nthreads*nblocks %i\n", nthreads, nblocks3, nthreads*nblocks3 );
  printf("per bin per segment grid: nthreads %i, nblocks %i nthreads*nblocks %i\n", nthreads, nblocks2, nthreads*nblocks2 );
  printf("per bin per fill grid: nthreads %i, nblocks %i nthreads*nblocks %i\n", nthreads, nblocks5, nthreads*nblocks5 );
  printf("per bin per segment per fill grid: nthreads %i, nblocks %i nthreads*nblocks %i\n", nthreads, nblocks4, nthreads*nblocks4 );

  // histogram binning
  printf("ns per fill %i, ns per bin %i, number of bins %d\n", nsPerFill, nsPerTick, fill_buffer_max_length);
  Char_t hname[256];
  for (int ih = 0; ih < nsegs; ih++) {

    for (int it = 0; it < nthresholds; it++) {

      //printf("hFlush1D: fill ih, it, ih + nthresholds*i) %i, %i, %i\n", ih, it, ih*nthresholds + it);

      sprintf( hname, "\n hFlush1D%02i_%02i", ih, it);
      hFlush1D[ih*nthresholds + it] = new TH1D( hname, hname, fill_buffer_max_length, 0.0, fill_buffer_max_length );
      sprintf( hname, "\n hFlush1Dlost%02i_%02i", ih, it);
      hFlush1Dlost[ih*nthresholds + it] = new TH1D( hname, hname, fill_buffer_max_length, 0.0, fill_buffer_max_length );
    }

    sprintf( hname, "\n hFlush2D%02i", ih);
    hFlush2D[ih] = new TH2D( hname, hname, fill_buffer_max_length, 0.0, fill_buffer_max_length, 64, -32, 31 );
    sprintf( hname, "\n hFlush2DCoarse%02i", ih);
    hFlush2DCoarse[ih] = new TH2D( hname, hname, fill_buffer_max_length, 0.0, fill_buffer_max_length, 256, 0, 8192 );
  }

  sprintf( hname, "\n hHits1D");
  hHits1D = new TH1D( hname, hname, fill_buffer_max_length, 0.0, fill_buffer_max_length );
  sprintf( hname, "\n hEnergy1D");
  hEnergy1D = new TH1D( hname, hname, energybins, 0.0, energybins );
  sprintf( hname, "\n hFlush2DSum");
  hFlush2DSum = new TH2D( hname, hname, fill_buffer_max_length, 0.0, fill_buffer_max_length, 64, -32, 31 );
  sprintf( hname, "\n hFlush2DCoarseSum");
  hFlush2DCoarseSum = new TH2D( hname, hname, fill_buffer_max_length, 0.0, fill_buffer_max_length, 256, 0, 8192 );


  // switch to do fill-by-fill noise
  bool fillbyfillnoise = false; 
  // switch to do flush-by-flush noise
  bool flushbyflushnoise = false; 

  // set device number for GPU
  int num_devices, device;
  hipGetDeviceCount(&num_devices);
  if (num_devices > 1) {
     for (device = 0; device < num_devices; device++) {
  	 hipDeviceProp_t properties;
	 hipGetDeviceProperties(&properties, device);
         printf("device %d properties.multiProcessorCount %d\n", device, properties.multiProcessorCount);
     }			      
  }	      
  hipSetDevice(0);

  // get some cuda device properties
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("Device Number: %d\n", 0);
  printf("Device name: %s\n", prop.name);
  printf("Memory Clock Rate (KHz): %d\n",
  		 prop.memoryClockRate);
  printf("Memory Bus Width (bits): %d\n",
                  prop.memoryBusWidth);


  // paramters for time measurements of GPU performance
  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  // device arrays for random number logic
  // could avoid the factor nfills in arrays by using atomicAdd() if only want flushed histograms?
  hipMalloc( (void **)&d_state, nsegs*nfills*sizeof(hiprandState));
  hipMalloc( (void **)&d_state2, nsegs*fill_buffer_max_length*sizeof(hiprandState));
  err = hipDeviceSynchronize();
  if ( hipSuccess != err ) {
    printf("Cuda error in file '%s' in line %i : %s.\n",
             __FILE__, __LINE__, hipGetErrorString( err) );
  }

  // host, device arrays for flushes
  //below are for flush-by-flush arrays  
  h_fillSumArray = (float *)malloc(nsegs*fill_buffer_max_length*sizeof(float));
  hipMalloc( (void **)&d_fillSumArray, nsegs*fill_buffer_max_length*sizeof(float));
  h_hitSumArray = (float *)malloc(fill_buffer_max_length*sizeof(float));
  hipMalloc( (void **)&d_hitSumArray, fill_buffer_max_length*sizeof(float));
  h_energySumArray = (float *)malloc(energybins*sizeof(float));
  hipMalloc( (void **)&d_energySumArray, energybins*sizeof(float));
  err = hipDeviceSynchronize();
  if ( hipSuccess != err ) {
    printf("Cuda error in file '%s' in line %i : %s.\n",
             __FILE__, __LINE__, hipGetErrorString( err) );
  }

  // measure time for array allocation
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel malloc / hipMalloc time %f ms\n",elapsedTime);

  // initialization for random number generator every 100 flushes
  hipEventRecord(start, 0);

  init_rand<<<nblocks1,nthreads>>>( d_state, 0, time(NULL));
  init_rand<<<nblocks2,nthreads>>>( d_state2, 0, time(NULL));
  
  // measure time for random number initialization
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" ::: kernel init_rand time %f ms\n",elapsedTime);


  struct timeval start_time, end_time;
  gettimeofday(&start_time, NULL);

  // loop over flushes in ruhn
  for (int j = 0; j < nflushes; j++){

    printf("flush %i\n", j);


    // initialize to zero the arrays storing fill, hits and energy
    hipEventRecord(start, 0);

    hipMemset( d_fillSumArray, 0.0, nsegs*fill_buffer_max_length*sizeof(float));
    if ( err != hipSuccess ) {
      printf("hipMemset error!\n");
      return 1;
    }
    hipMemset( d_hitSumArray, 0.0, fill_buffer_max_length*sizeof(float));
    if ( err != hipSuccess ) {
      printf("hipMemset error!\n");
      return 1;
    }
    hipMemset( d_energySumArray, 0.0, energybins*sizeof(float));
    if ( err != hipSuccess ) {
      printf("hipMemset error!\n");
      return 1;
    }

    hipDeviceSynchronize();

    // measure time for array initialization
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    if (j == 0) printf(" ::: kernel initialize fillArray, hitArray, energyArray %f ms\n",elapsedTime);

    // make the fills within the flush
    hipEventRecord(start, 0);
    make_randfill<<<nblocks1,nthreads>>>( d_state, d_hitArray, d_fillArray, d_hitSumArray, d_fillSumArray, 
					  d_energySumArray, ne, fill_buffer_max_length, nfills, fillbyfillnoise);
    err=hipGetLastError();
    if(err!=hipSuccess) {
      printf("Cuda failure with user kernel function %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(err));
      exit(0);
    } 

    // measure time for making fliss with single flush
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    if (j == 0) printf(" ::: kernel make_randfill time %f ms\n",elapsedTime);

    // copy the flush from GPU to CPU
    hipEventRecord(start, 0);
    hipMemcpy( h_fillSumArray, d_fillSumArray, nsegs*fill_buffer_max_length*sizeof(float), hipMemcpyDeviceToHost);
    err=hipGetLastError();
    if(err!=hipSuccess) {
      printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(err));
      exit(0);
    }  
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventRecord(start, 0);
    hipMemcpy( h_hitSumArray, d_hitSumArray, fill_buffer_max_length*sizeof(float), hipMemcpyDeviceToHost);
    err=hipGetLastError();
    if(err!=hipSuccess) {
      printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(err));
      exit(0);
    }  
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventRecord(start, 0);
    hipMemcpy( h_energySumArray, d_energySumArray, energybins*sizeof(float), hipMemcpyDeviceToHost);
    err=hipGetLastError();
    if(err!=hipSuccess) {
      printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(err));
      exit(0);
    }  
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // measure time for copying the flush from GPU to  CPU
    hipEventElapsedTime(&elapsedTime, start, stop);
    if (j == 0) printf(" ::: kernel cuda memcpy time for flush array ms\n",elapsedTime);

    // add flush to run root histograms
    for (int i = 0; i < nsegs*fill_buffer_max_length; i++){

      int ih = i / fill_buffer_max_length;
      int ib = i % fill_buffer_max_length;
      //if (ib == 0) printf("i %i, ib %i, ih %i, *(h_fillSumArray+i) %f\n", i, ib, ih, *(h_fillSumArray+i));

      for (int it = 0; it < nthresholds; it++) {
	
	//if (ib == 0) printf("hFlush1D: fill ih, it, ih + nthresholds*i) %i, %i, %i\n", ih, it, ih*nthresholds + it);

	if ( *(h_fillSumArray+i) >= it*threshold) {
	  hFlush1D[ih*nthresholds + it]->Fill( ib+1, *(h_fillSumArray+i));
	} else {
	  hFlush1Dlost[ih*nthresholds + it]->Fill( ib+1, *(h_fillSumArray+i));
	}
      }

      hFlush2D[ih]->Fill( ib+1, *(h_fillSumArray+i));
      hFlush2DCoarse[ih]->Fill( ib+1, *(h_fillSumArray+i));
      //fprintf(fp, " %i %f\n", i+1, *(h_fillSumArray+i) );      
    }

    // make xtal-summmed distributions
    for (int ib = 0; ib < fill_buffer_max_length; ib++){

      float sum = 0.0;
      for (int is = 0; is < nsegs; is++){
	sum += *(h_fillSumArray + is*fill_buffer_max_length + ib);
      }

      hFlush2DSum->Fill( ib+1, sum);
      hFlush2DCoarseSum->Fill( ib+1, sum);
    }
  
    // fill diagnostic hit distribution
    for (int ib = 0; ib < fill_buffer_max_length; ib++){
      hHits1D->Fill( ib+1, *(h_hitSumArray+ib));
    }

    // fill diagnostic energy distribution
    for (int ib = 0; ib < energybins; ib++){
      hEnergy1D->Fill( ib+1, *(h_energySumArray+ib));
    }

  }

  // free device arrays
  hipFree(d_state);
  hipFree(d_state2);
  hipFree(d_fillSumArray);
  hipFree(d_hitSumArray);
  hipFree(d_energySumArray);

  // time elapsed for gnerating the entire run 
  gettimeofday(&end_time, NULL);
  printf("elapsed processing time, dt %f secs\n", toddiff(&end_time, &start_time));

  // open root file and write root hostograms
  Char_t fname[256];
  sprintf( fname, "root/threshold-qmethod-thres%03i-ne%05i-nfill%05i-nflush%05i.root", (int)threshold, ne, nfills, nflushes);
  f = new TFile(fname,"recreate");
  printf("write histograms\n"); 

  for (int ih = 0; ih < nsegs; ih++) {
    printf("writing segment %i\n", ih);

    for (int it = 0; it < nthresholds; it++) {
      
      sprintf( hname, "h%02i_%02i", ih, it);
      f->WriteObject( hFlush1D[ih*nthresholds + it], hname);
      sprintf( hname, "hlost%02i_%02i", ih, it);
      f->WriteObject( hFlush1Dlost[ih*nthresholds + it], hname);
    }

    sprintf( hname, "s%02i", ih);
    f->WriteObject( hFlush2D[ih], hname);
    sprintf( hname, "sc%02i", ih);
    f->WriteObject( hFlush2DCoarse[ih], hname);
  }

  sprintf( hname, "hHits");
  f->WriteObject( hHits1D, hname);
  sprintf( hname, "hEnergy");
  f->WriteObject( hEnergy1D, hname);
  sprintf( hname, "sSum");
  f->WriteObject( hFlush2DSum, hname);
  sprintf( hname, "scSum");
  f->WriteObject( hFlush2DCoarseSum, hname);
  f->Close();

  return 0;
}

